#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            return; \
        } \
    } while(0)

// Optimized SMA kernel using shared memory for small/medium periods
__global__ void sma_kernel(const double* prices, double* sma, int n, int period) {
    extern __shared__ double s_prices[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if (idx >= n) return;

    // Load data into shared memory (window for this block)
    int window_start = idx - period + 1;
    int window_end = idx;
    for (int i = 0; i < period; ++i) {
        int data_idx = window_start + i;
        if (data_idx >= 0 && data_idx < n) {
            s_prices[tid * period + i] = prices[data_idx];
        } else {
            s_prices[tid * period + i] = 0.0;
        }
    }
    __syncthreads();

    if (idx < period - 1) {
        sma[idx] = 0.0;
        return;
    }

    double sum = 0.0;
    for (int i = 0; i < period; ++i) {
        sum += s_prices[tid * period + i];
    }
    sma[idx] = sum / period;
}

// Optimized RSI kernel using shared memory for small/medium periods
__global__ void rsi_kernel(const double* prices, double* rsi, int n, int period) {
    extern __shared__ double s_prices[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if (idx >= n) return;

    // Load data into shared memory (window for this block)
    int window_start = idx - period + 1;
    for (int i = 0; i <= period; ++i) {
        int data_idx = window_start + i - 1;
        if (data_idx >= 0 && data_idx < n) {
            s_prices[tid * (period + 1) + i] = prices[data_idx];
        } else {
            s_prices[tid * (period + 1) + i] = 0.0;
        }
    }
    __syncthreads();

    if (idx < period) {
        rsi[idx] = 50.0;
        return;
    }

    double gain = 0.0, loss = 0.0;
    for (int i = 1; i <= period; ++i) {
        double change = s_prices[tid * (period + 1) + i] - s_prices[tid * (period + 1) + i - 1];
        if (change > 0) gain += change;
        else loss -= change;
    }

    if (gain + loss == 0) {
        rsi[idx] = 50.0;
    } else {
        double rs = gain / (loss == 0 ? 1e-10 : loss);
        rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
    }
}

// GPU kernel for signal generation
__global__ void signal_kernel(
    const double* prices, const double* sma, const double* rsi,
    int* signals, double* stops, double* targets,
    int n, double rsi_oversold, double risk_reward
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    // Check conditions for buy signal
    bool uptrend = prices[idx] > sma[idx];
    bool oversold = rsi[idx] < rsi_oversold;
    
    // Simple FVG detection (gap up/down)
    bool fvg = false;
    if (idx >= 1) {
        fvg = (prices[idx] > prices[idx-1] * 1.01) || (prices[idx] < prices[idx-1] * 0.99);
    }
    
    if (uptrend && oversold && fvg) {
        signals[idx] = 1; // BUY signal
        double entry = prices[idx];
        double stop_loss_pct = 0.005 / risk_reward;
        stops[idx] = entry - (entry * stop_loss_pct);
        targets[idx] = entry + (entry - stops[idx]) * risk_reward;
    } else {
        signals[idx] = 0; // NO signal
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Host wrapper functions
extern "C" {

void gpu_calculate_indicators(
    const double* prices, int n,
    double* sma, double* rsi,
    int sma_period, int rsi_period
) {
    // Allocate device memory
    double *d_prices, *d_sma, *d_rsi;
    CUDA_CHECK(hipMalloc(&d_prices, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_sma, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_rsi, n * sizeof(double)));
    
    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice));
    
    // Launch kernels with shared memory
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    size_t sma_shmem = block_size * sma_period * sizeof(double);
    size_t rsi_shmem = block_size * (rsi_period + 1) * sizeof(double);
    
    sma_kernel<<<grid_size, block_size, sma_shmem>>>(d_prices, d_sma, n, sma_period);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "SMA kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    rsi_kernel<<<grid_size, block_size, rsi_shmem>>>(d_prices, d_rsi, n, rsi_period);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "RSI kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Copy results back
    CUDA_CHECK(hipMemcpy(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost));
    
    // Cleanup
    hipFree(d_prices);
    hipFree(d_sma);
    hipFree(d_rsi);
}

void gpu_generate_signals(
    const double* prices, const double* sma, const double* rsi,
    int n, double rsi_oversold, double risk_reward,
    int* signals, double* stops, double* targets
) {
    // Allocate device memory
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    CUDA_CHECK(hipMalloc(&d_prices, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_sma, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_rsi, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_signals, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_stops, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_targets, n * sizeof(double)));
    
    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sma, sma, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rsi, rsi, n * sizeof(double), hipMemcpyHostToDevice));
    
    // Launch kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    signal_kernel<<<grid_size, block_size>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, rsi_oversold, risk_reward
    );
    
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Signal kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Copy results back
    CUDA_CHECK(hipMemcpy(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost));
    
    // Cleanup
    hipFree(d_prices);
    hipFree(d_sma);
    hipFree(d_rsi);
    hipFree(d_signals);
    hipFree(d_stops);
    hipFree(d_targets);
}

} // extern "C" 
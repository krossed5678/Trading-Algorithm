#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            return; \
        } \
    } while(0)

__global__ void sma_kernel(const double* prices, double* sma, int n, int period) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    if (idx < period - 1) {
        sma[idx] = 0.0;
        return;
    }
    
    double sum = 0.0;
    for (int i = idx - period + 1; i <= idx; i++) {
        sum += prices[i];
    }
    sma[idx] = sum / period;
}

__global__ void rsi_kernel(const double* prices, double* rsi, int n, int period) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    if (idx < period) {
        rsi[idx] = 50.0;
        return;
    }
    
    double gain = 0.0, loss = 0.0;
    for (int i = idx - period + 1; i <= idx; i++) {
        double change = prices[i] - prices[i - 1];
        if (change > 0) gain += change;
        else loss -= change;
    }
    
    if (gain + loss == 0) {
        rsi[idx] = 50.0;
    } else {
        double rs = gain / (loss == 0 ? 1e-10 : loss);
        rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
    }
}

// GPU kernel for signal generation
__global__ void signal_kernel(
    const double* prices, const double* sma, const double* rsi,
    int* signals, double* stops, double* targets,
    int n, double rsi_oversold, double risk_reward
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    // Check conditions for buy signal
    bool uptrend = prices[idx] > sma[idx];
    bool oversold = rsi[idx] < rsi_oversold;
    
    // Simple FVG detection (gap up/down)
    bool fvg = false;
    if (idx >= 1) {
        fvg = (prices[idx] > prices[idx-1] * 1.01) || (prices[idx] < prices[idx-1] * 0.99);
    }
    
    if (uptrend && oversold && fvg) {
        signals[idx] = 1; // BUY signal
        double entry = prices[idx];
        double stop_loss_pct = 0.005 / risk_reward;
        stops[idx] = entry - (entry * stop_loss_pct);
        targets[idx] = entry + (entry - stops[idx]) * risk_reward;
    } else {
        signals[idx] = 0; // NO signal
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Host wrapper functions
extern "C" {

void gpu_calculate_indicators(
    const double* prices, int n,
    double* sma, double* rsi,
    int sma_period, int rsi_period
) {
    // Allocate device memory
    double *d_prices, *d_sma, *d_rsi;
    CUDA_CHECK(hipMalloc(&d_prices, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_sma, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_rsi, n * sizeof(double)));
    
    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice));
    
    // Launch kernels
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    sma_kernel<<<grid_size, block_size>>>(d_prices, d_sma, n, sma_period);
    rsi_kernel<<<grid_size, block_size>>>(d_prices, d_rsi, n, rsi_period);
    
    // Copy results back
    CUDA_CHECK(hipMemcpy(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost));
    
    // Cleanup
    hipFree(d_prices);
    hipFree(d_sma);
    hipFree(d_rsi);
}

void gpu_generate_signals(
    const double* prices, const double* sma, const double* rsi,
    int n, double rsi_oversold, double risk_reward,
    int* signals, double* stops, double* targets
) {
    // Allocate device memory
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    CUDA_CHECK(hipMalloc(&d_prices, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_sma, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_rsi, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_signals, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_stops, n * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_targets, n * sizeof(double)));
    
    // Copy data to GPU
    CUDA_CHECK(hipMemcpy(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sma, sma, n * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rsi, rsi, n * sizeof(double), hipMemcpyHostToDevice));
    
    // Launch kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    signal_kernel<<<grid_size, block_size>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, rsi_oversold, risk_reward
    );
    
    // Copy results back
    CUDA_CHECK(hipMemcpy(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost));
    
    // Cleanup
    hipFree(d_prices);
    hipFree(d_sma);
    hipFree(d_rsi);
    hipFree(d_signals);
    hipFree(d_stops);
    hipFree(d_targets);
}

} // extern "C" 
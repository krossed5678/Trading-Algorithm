#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            return; \
        } \
    } while(0)

// Optimized block size calculation
__device__ __forceinline__ int getOptimalBlockSize() {
    return 256; // Optimized for most modern GPUs
}

// Optimized SMA kernel with improved memory coalescing
__global__ void optimized_sma_kernel(const double* prices, double* sma, int n, int period) {
    extern __shared__ double s_prices[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx >= n) return;

    // Coalesced memory access pattern
    int window_start = max(0, idx - period + 1);
    int window_end = min(n - 1, idx);
    int window_size = window_end - window_start + 1;
    
    // Load data into shared memory with coalesced access
    double local_sum = 0.0;
    for (int i = window_start; i <= window_end; i += blockDim.x) {
        int load_idx = i + tid;
        if (load_idx <= window_end) {
            local_sum += prices[load_idx];
        }
    }
    
    // Reduce within thread block
    __shared__ double s_sums[256];
    s_sums[tid] = local_sum;
    __syncthreads();
    
    // Parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_sums[tid] += s_sums[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        int block_start = blockIdx.x * blockDim.x;
        int block_end = min(n - 1, block_start + blockDim.x - 1);
        int actual_period = min(period, block_end - block_start + 1);
        sma[idx] = s_sums[0] / actual_period;
    }
}

// Optimized RSI kernel with improved numerical stability
__global__ void optimized_rsi_kernel(const double* prices, double* rsi, int n, int period) {
    extern __shared__ double s_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx >= n) return;
    
    if (idx < period) {
        rsi[idx] = 50.0;
        return;
    }
    
    // Load price data into shared memory
    int start_idx = idx - period;
    for (int i = 0; i <= period; i += blockDim.x) {
        int load_idx = start_idx + i + tid;
        if (load_idx <= idx && load_idx < n) {
            s_data[tid * (period + 1) + i] = prices[load_idx];
        }
    }
    __syncthreads();
    
    // Calculate gains and losses
    double gain_sum = 0.0, loss_sum = 0.0;
    for (int i = 1; i <= period; i++) {
        double change = s_data[tid * (period + 1) + i] - s_data[tid * (period + 1) + i - 1];
        if (change > 0) {
            gain_sum += change;
        } else if (change < 0) {
            loss_sum -= change;
        }
    }
    
    // Improved numerical stability
    if (gain_sum + loss_sum < 1e-10) {
        rsi[idx] = 50.0;
    } else {
        double avg_gain = gain_sum / period;
        double avg_loss = loss_sum / period;
        
        if (avg_loss < 1e-10) {
            rsi[idx] = 100.0;
        } else {
            double rs = avg_gain / avg_loss;
            rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
        }
    }
}

// Fused kernel that calculates SMA, RSI, and generates signals in one pass
__global__ void fused_indicators_kernel(
    const double* prices, 
    double* sma, double* rsi, int* signals, double* stops, double* targets,
    int n, int sma_period, int rsi_period, double rsi_oversold, double risk_reward
) {
    extern __shared__ double s_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx >= n) return;
    
    // Calculate SMA
    if (idx >= sma_period - 1) {
        double sma_sum = 0.0;
        for (int i = idx - sma_period + 1; i <= idx; i++) {
            sma_sum += prices[i];
        }
        sma[idx] = sma_sum / sma_period;
    } else {
        sma[idx] = 0.0;
    }
    
    // Calculate RSI
    if (idx >= rsi_period) {
        double gain_sum = 0.0, loss_sum = 0.0;
        for (int i = idx - rsi_period + 1; i <= idx; i++) {
            double change = prices[i] - prices[i - 1];
            if (change > 0) gain_sum += change;
            else loss_sum -= change;
        }
        
        if (gain_sum + loss_sum < 1e-10) {
            rsi[idx] = 50.0;
        } else {
            double avg_gain = gain_sum / rsi_period;
            double avg_loss = loss_sum / rsi_period;
            
            if (avg_loss < 1e-10) {
                rsi[idx] = 100.0;
            } else {
                double rs = avg_gain / avg_loss;
                rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
            }
        }
    } else {
        rsi[idx] = 50.0;
    }
    
    // Generate signals
    if (idx >= max(sma_period, rsi_period)) {
        bool uptrend = prices[idx] > sma[idx];
        bool oversold = rsi[idx] < rsi_oversold;
        
        // Enhanced FVG detection
        bool fvg = false;
        if (idx >= 1) {
            double gap_threshold = 0.01; // 1% threshold
            fvg = (prices[idx] > prices[idx-1] * (1.0 + gap_threshold)) || 
                  (prices[idx] < prices[idx-1] * (1.0 - gap_threshold));
        }
        
        if (uptrend && oversold && fvg) {
            signals[idx] = 1; // BUY signal
            double entry = prices[idx];
            double stop_loss_pct = 0.005 / risk_reward;
            stops[idx] = entry * (1.0 - stop_loss_pct);
            targets[idx] = entry + (entry - stops[idx]) * risk_reward;
        } else {
            signals[idx] = 0; // NO signal
            stops[idx] = 0.0;
            targets[idx] = 0.0;
        }
    } else {
        signals[idx] = 0;
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Optimized signal generation kernel with vectorized operations
__global__ void optimized_signal_kernel(
    const double* prices, const double* sma, const double* rsi,
    int* signals, double* stops, double* targets,
    int n, double rsi_oversold, double risk_reward
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    // Vectorized condition checking
    double price = prices[idx];
    double sma_val = sma[idx];
    double rsi_val = rsi[idx];
    
    // Use bitwise operations for condition checking
    bool uptrend = price > sma_val;
    bool oversold = rsi_val < rsi_oversold;
    
    // Enhanced FVG detection with multiple timeframes
    bool fvg = false;
    if (idx >= 1) {
        double gap_threshold = 0.01;
        double price_change = price / prices[idx-1];
        fvg = (price_change > 1.0 + gap_threshold) || (price_change < 1.0 - gap_threshold);
    }
    
    // Combined condition check
    if (uptrend && oversold && fvg) {
        signals[idx] = 1;
        double entry = price;
        double stop_loss_pct = 0.005 / risk_reward;
        stops[idx] = entry * (1.0 - stop_loss_pct);
        targets[idx] = entry + (entry - stops[idx]) * risk_reward;
    } else {
        signals[idx] = 0;
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Memory pool for reducing allocation overhead
class CUDAMemoryPool {
private:
    struct MemoryBlock {
        void* ptr;
        size_t size;
        bool in_use;
    };
    std::vector<MemoryBlock> blocks_;
    
public:
    void* allocate(size_t size) {
        // Find existing block
        for (auto& block : blocks_) {
            if (!block.in_use && block.size >= size) {
                block.in_use = true;
                return block.ptr;
            }
        }
        
        // Allocate new block
        void* ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        blocks_.push_back({ptr, size, true});
        return ptr;
    }
    
    void free(void* ptr) {
        for (auto& block : blocks_) {
            if (block.ptr == ptr) {
                block.in_use = false;
                return;
            }
        }
    }
    
    ~CUDAMemoryPool() {
        for (auto& block : blocks_) {
            hipFree(block.ptr);
        }
    }
};

static CUDAMemoryPool g_memory_pool;

// Host wrapper functions with optimized memory management
extern "C" {

void gpu_calculate_indicators(
    const double* prices, int n,
    double* sma, double* rsi,
    int sma_period, int rsi_period
) {
    // Use fused kernel for better performance
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    // Allocate device memory using pool
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Asynchronous memory copy
    CUDA_CHECK(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch fused kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    size_t shared_mem = block_size * max(sma_period, rsi_period + 1) * sizeof(double);
    
    fused_indicators_kernel<<<grid_size, block_size, shared_mem>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, sma_period, rsi_period, 30.0, 2.0
    );
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Fused kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Asynchronous copy back
    CUDA_CHECK(hipMemcpyAsync(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    // Synchronize
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

void gpu_generate_signals(
    const double* prices, const double* sma, const double* rsi,
    int n, double rsi_oversold, double risk_reward,
    int* signals, double* stops, double* targets
) {
    // Allocate device memory using pool
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Asynchronous memory copies
    CUDA_CHECK(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    CUDA_CHECK(hipMemcpyAsync(d_sma, sma, n * sizeof(double), hipMemcpyHostToDevice, 0));
    CUDA_CHECK(hipMemcpyAsync(d_rsi, rsi, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch optimized signal kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    optimized_signal_kernel<<<grid_size, block_size>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, rsi_oversold, risk_reward
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Signal kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Asynchronous copy back
    CUDA_CHECK(hipMemcpyAsync(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    // Synchronize
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

// New optimized function that does everything in one GPU call
void gpu_calculate_all_indicators_and_signals(
    const double* prices, int n,
    double* sma, double* rsi, int* signals, double* stops, double* targets,
    int sma_period, int rsi_period, double rsi_oversold, double risk_reward
) {
    // Allocate device memory using pool
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Single memory copy
    CUDA_CHECK(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch single fused kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    size_t shared_mem = block_size * max(sma_period, rsi_period + 1) * sizeof(double);
    
    fused_indicators_kernel<<<grid_size, block_size, shared_mem>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, sma_period, rsi_period, rsi_oversold, risk_reward
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Fused kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Single synchronization and copy back
    CUDA_CHECK(hipMemcpyAsync(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK(hipMemcpyAsync(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

} // extern "C" 
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

// Error checking macros
#define CUDA_CHECK_RETURN_NULLPTR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            return nullptr; \
        } \
    } while(0)
#define CUDA_CHECK_RETURN_VOID(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            return; \
        } \
    } while(0)

// Optimized block size calculation
__device__ __forceinline__ int getOptimalBlockSize() {
    return 256; // Optimized for most modern GPUs
}

// Optimized SMA kernel with improved memory coalescing
__global__ void optimized_sma_kernel(const double* prices, double* sma, int n, int period) {
    extern __shared__ double s_prices[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx >= n) return;

    // Coalesced memory access pattern
    int window_start = max(0, idx - period + 1);
    int window_end = min(n - 1, idx);
    
    // Load data into shared memory with coalesced access
    double local_sum = 0.0;
    for (int i = window_start; i <= window_end; i += blockDim.x) {
        int load_idx = i + tid;
        if (load_idx <= window_end) {
            local_sum += prices[load_idx];
        }
    }
    
    // Reduce within thread block
    __shared__ double s_sums[256];
    s_sums[tid] = local_sum;
    __syncthreads();
    
    // Parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_sums[tid] += s_sums[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        int block_start = blockIdx.x * blockDim.x;
        int block_end = min(n - 1, block_start + blockDim.x - 1);
        int actual_period = min(period, block_end - block_start + 1);
        sma[idx] = s_sums[0] / actual_period;
    }
}

// Optimized RSI kernel with improved numerical stability
__global__ void optimized_rsi_kernel(const double* prices, double* rsi, int n, int period) {
    extern __shared__ double s_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n) return;
    
    if (idx < period) {
        rsi[idx] = 50.0;
        return;
    }
    
    // Load price data into shared memory
    int start_idx = idx - period;
    for (int i = 0; i <= period; i += blockDim.x) {
        int load_idx = start_idx + i + threadIdx.x;
        if (load_idx <= idx && load_idx < n) {
            s_data[threadIdx.x * (period + 1) + i] = prices[load_idx];
        }
    }
    __syncthreads();
    
    // Calculate gains and losses
    double gain_sum = 0.0, loss_sum = 0.0;
    for (int i = 1; i <= period; i++) {
        double change = s_data[threadIdx.x * (period + 1) + i] - s_data[threadIdx.x * (period + 1) + i - 1];
        if (change > 0) {
            gain_sum += change;
        } else if (change < 0) {
            loss_sum -= change;
        }
    }
    
    // Improved numerical stability
    if (gain_sum + loss_sum < 1e-10) {
        rsi[idx] = 50.0;
    } else {
        double avg_gain = gain_sum / period;
        double avg_loss = loss_sum / period;
        
        if (avg_loss < 1e-10) {
            rsi[idx] = 100.0;
        } else {
            double rs = avg_gain / avg_loss;
            rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
        }
    }
}

// Fused kernel that calculates SMA, RSI, and generates signals in one pass
__global__ void fused_indicators_kernel(
    const double* prices, 
    double* sma, double* rsi, int* signals, double* stops, double* targets,
    int n, int sma_period, int rsi_period, double rsi_oversold, double risk_reward
) {
    extern __shared__ double s_data[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (idx >= n) return;
    
    // Calculate SMA
    if (idx >= sma_period - 1) {
        double sma_sum = 0.0;
        for (int i = idx - sma_period + 1; i <= idx; i++) {
            sma_sum += prices[i];
        }
        sma[idx] = sma_sum / sma_period;
    } else {
        sma[idx] = 0.0;
    }
    
    // Calculate RSI
    if (idx >= rsi_period) {
        double gain_sum = 0.0, loss_sum = 0.0;
        for (int i = idx - rsi_period + 1; i <= idx; i++) {
            double change = prices[i] - prices[i - 1];
            if (change > 0) gain_sum += change;
            else loss_sum -= change;
        }
        
        if (gain_sum + loss_sum < 1e-10) {
            rsi[idx] = 50.0;
        } else {
            double avg_gain = gain_sum / rsi_period;
            double avg_loss = loss_sum / rsi_period;
            
            if (avg_loss < 1e-10) {
                rsi[idx] = 100.0;
            } else {
                double rs = avg_gain / avg_loss;
                rsi[idx] = 100.0 - (100.0 / (1.0 + rs));
            }
        }
    } else {
        rsi[idx] = 50.0;
    }
    
    // Generate signals
    if (idx >= max(sma_period, rsi_period)) {
        bool uptrend = prices[idx] > sma[idx];
        bool oversold = rsi[idx] < rsi_oversold;
        
        // Enhanced FVG detection
        bool fvg = false;
        if (idx >= 1) {
            double gap_threshold = 0.01; // 1% threshold
            fvg = (prices[idx] > prices[idx-1] * (1.0 + gap_threshold)) || 
                  (prices[idx] < prices[idx-1] * (1.0 - gap_threshold));
        }
        
        if (uptrend && oversold && fvg) {
            signals[idx] = 1; // BUY signal
            double entry = prices[idx];
            double stop_loss_pct = 0.005 / risk_reward;
            stops[idx] = entry * (1.0 - stop_loss_pct);
            targets[idx] = entry + (entry - stops[idx]) * risk_reward;
        } else {
            signals[idx] = 0; // NO signal
            stops[idx] = 0.0;
            targets[idx] = 0.0;
        }
    } else {
        signals[idx] = 0;
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Optimized signal generation kernel with vectorized operations
__global__ void optimized_signal_kernel(
    const double* prices, const double* sma, const double* rsi,
    int* signals, double* stops, double* targets,
    int n, double rsi_oversold, double risk_reward
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    
    // Vectorized condition checking
    double price = prices[idx];
    double sma_val = sma[idx];
    double rsi_val = rsi[idx];
    
    // Use bitwise operations for condition checking
    bool uptrend = price > sma_val;
    bool oversold = rsi_val < rsi_oversold;
    
    // Enhanced FVG detection with multiple timeframes
    bool fvg = false;
    if (idx >= 1) {
        double gap_threshold = 0.01;
        double price_change = price / prices[idx-1];
        fvg = (price_change > 1.0 + gap_threshold) || (price_change < 1.0 - gap_threshold);
    }
    
    // Combined condition check
    if (uptrend && oversold && fvg) {
        signals[idx] = 1;
        double entry = price;
        double stop_loss_pct = 0.005 / risk_reward;
        stops[idx] = entry * (1.0 - stop_loss_pct);
        targets[idx] = entry + (entry - stops[idx]) * risk_reward;
    } else {
        signals[idx] = 0;
        stops[idx] = 0.0;
        targets[idx] = 0.0;
    }
}

// Memory pool for reducing allocation overhead
class CUDAMemoryPool {
private:
    struct MemoryBlock {
        void* ptr;
        size_t size;
        bool in_use;
    };
    std::vector<MemoryBlock> blocks_;
    
public:
    void* allocate(size_t size) {
        // Find existing block
        for (auto& block : blocks_) {
            if (!block.in_use && block.size >= size) {
                block.in_use = true;
                return block.ptr;
            }
        }
        
        // Allocate new block
        void* ptr = nullptr;
        CUDA_CHECK_RETURN_NULLPTR(hipMalloc(&ptr, size));
        if (ptr != nullptr) {
            blocks_.push_back({ptr, size, true});
        }
        return ptr;
    }
    
    void free(void* ptr) {
        for (auto& block : blocks_) {
            if (block.ptr == ptr) {
                block.in_use = false;
                return;
            }
        }
    }
    
    ~CUDAMemoryPool() {
        for (auto& block : blocks_) {
            hipFree(block.ptr);
        }
    }
};

static CUDAMemoryPool g_memory_pool;

// Host wrapper functions with optimized memory management
extern "C" {

void gpu_calculate_indicators(
    const double* prices, int n,
    double* sma, double* rsi,
    int sma_period, int rsi_period
) {
    // Use fused kernel for better performance
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    // Allocate device memory using pool
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Asynchronous memory copy
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch fused kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    size_t shared_mem = block_size * max(sma_period, rsi_period + 1) * sizeof(double);
    
    fused_indicators_kernel<<<grid_size, block_size, shared_mem>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, sma_period, rsi_period, 30.0, 2.0
    );
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Fused kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Asynchronous copy back
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    // Synchronize
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

void gpu_generate_signals(
    const double* prices, const double* sma, const double* rsi,
    int n, double rsi_oversold, double risk_reward,
    int* signals, double* stops, double* targets
) {
    // Allocate device memory using pool
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Asynchronous memory copies
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(d_sma, sma, n * sizeof(double), hipMemcpyHostToDevice, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(d_rsi, rsi, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch optimized signal kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    
    optimized_signal_kernel<<<grid_size, block_size>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, rsi_oversold, risk_reward
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Signal kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Asynchronous copy back
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    // Synchronize
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

// New optimized function that does everything in one GPU call
void gpu_calculate_all_indicators_and_signals(
    const double* prices, int n,
    double* sma, double* rsi, int* signals, double* stops, double* targets,
    int sma_period, int rsi_period, double rsi_oversold, double risk_reward
) {
    // Allocate device memory using pool
    double *d_prices, *d_sma, *d_rsi;
    int *d_signals;
    double *d_stops, *d_targets;
    
    d_prices = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_sma = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_rsi = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_signals = (int*)g_memory_pool.allocate(n * sizeof(int));
    d_stops = (double*)g_memory_pool.allocate(n * sizeof(double));
    d_targets = (double*)g_memory_pool.allocate(n * sizeof(double));
    
    // Single memory copy
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(d_prices, prices, n * sizeof(double), hipMemcpyHostToDevice, 0));
    
    // Launch single fused kernel
    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;
    size_t shared_mem = block_size * max(sma_period, rsi_period + 1) * sizeof(double);
    
    fused_indicators_kernel<<<grid_size, block_size, shared_mem>>>(
        d_prices, d_sma, d_rsi, d_signals, d_stops, d_targets,
        n, sma_period, rsi_period, rsi_oversold, risk_reward
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Fused kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Single synchronization and copy back
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(sma, d_sma, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(rsi, d_rsi, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(signals, d_signals, n * sizeof(int), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(stops, d_stops, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    CUDA_CHECK_RETURN_VOID(hipMemcpyAsync(targets, d_targets, n * sizeof(double), hipMemcpyDeviceToHost, 0));
    
    hipDeviceSynchronize();
    
    // Free memory back to pool
    g_memory_pool.free(d_prices);
    g_memory_pool.free(d_sma);
    g_memory_pool.free(d_rsi);
    g_memory_pool.free(d_signals);
    g_memory_pool.free(d_stops);
    g_memory_pool.free(d_targets);
}

} // extern "C" 

// CUDA-compatible versions of the structs
struct CudaOHLCV {
    double open, high, low, close, volume;
};

struct CudaStrategyGene {
    int primary_indicator;
    int secondary_indicator;
    int primary_period;
    int secondary_period;
    double primary_threshold;
    double secondary_threshold;
    int entry_condition;
    int exit_condition;
    double risk_reward_ratio;
    double stop_loss_pct;
    double take_profit_pct;
    int max_hold_time;
    double position_size_pct;
};

struct CudaFitnessResult {
    double total_return;
    double sharpe_ratio;
    double max_drawdown;
    double win_rate;
    int total_trades;
    double profit_factor;
    double calmar_ratio;
    double fitness_score;
};

__device__ double cuda_max(double a, double b) { return a > b ? a : b; }
__device__ double cuda_min(double a, double b) { return a < b ? a : b; }

__device__ double calculateSharpeRatio(const double* returns, int n) {
    if (n == 0) return 0.0;
    double mean = 0.0;
    for (int i = 0; i < n; ++i) mean += returns[i];
    mean /= n;
    double variance = 0.0;
    for (int i = 0; i < n; ++i) variance += (returns[i] - mean) * (returns[i] - mean);
    variance /= n;
    double std_dev = sqrt(variance);
    return (std_dev > 0) ? mean / std_dev : 0.0;
}

__device__ double calculateMaxDrawdown(const double* equity_curve, int n) {
    if (n == 0) return 0.0;
    double max_dd = 0.0, peak = equity_curve[0];
    for (int i = 0; i < n; ++i) {
        if (equity_curve[i] > peak) peak = equity_curve[i];
        double dd = (peak - equity_curve[i]) / peak;
        if (dd > max_dd) max_dd = dd;
    }
    return max_dd;
}

__device__ double calculateProfitFactor(const double* profits, int n_profits, const double* losses, int n_losses) {
    double total_profit = 0.0, total_loss = 0.0;
    for (int i = 0; i < n_profits; ++i) total_profit += profits[i];
    for (int i = 0; i < n_losses; ++i) total_loss += losses[i];
    return (total_loss > 0) ? total_profit / total_loss : (total_profit > 0) ? 1000.0 : 0.0;
}

__global__ void evaluate_population_kernel(
    const CudaStrategyGene* genes,
    const CudaOHLCV* data,
    int population_size,
    int data_size,
    CudaFitnessResult* results
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= population_size) return;
    const CudaStrategyGene& gene = genes[idx];
    double current_equity = 10000.0;
    int winning_trades = 0, total_trades = 0;
    double equity_curve[1024]; // Assume max 1024 bars for demo; for real use, use dynamic alloc or limit size
    double returns[1024];
    double profits[256], losses[256];
    int n_profits = 0, n_losses = 0;
    int eq_idx = 0, ret_idx = 0;
    for (int i = 0; i < data_size; ++i) {
        // Simple buy signal: if close > open (placeholder for real logic)
        if (data[i].close > data[i].open) {
            double entry_price = data[i].close;
            double stop_loss = entry_price * (1.0 - gene.stop_loss_pct);
            double take_profit = entry_price * (1.0 + gene.take_profit_pct);
            for (int j = i + 1; j < data_size; ++j) {
                if (data[j].low <= stop_loss || data[j].high >= take_profit) {
                    double exit_price = (data[j].low <= stop_loss) ? stop_loss : take_profit;
                    double trade_return = (exit_price - entry_price) / entry_price;
                    if (trade_return > 0) { winning_trades++; profits[n_profits++] = trade_return; }
                    else { losses[n_losses++] = -trade_return; }
                    total_trades++;
                    current_equity *= (1 + trade_return * gene.position_size_pct);
                    break;
                }
            }
        }
        equity_curve[eq_idx++] = current_equity;
        if (i > 0) returns[ret_idx++] = (current_equity - equity_curve[i-1]) / equity_curve[i-1];
    }
    CudaFitnessResult result;
    result.total_return = (current_equity - 10000.0) / 10000.0;
    result.sharpe_ratio = calculateSharpeRatio(returns, ret_idx);
    result.max_drawdown = calculateMaxDrawdown(equity_curve, eq_idx);
    result.win_rate = (total_trades > 0) ? ((double)winning_trades / total_trades) : 0.0;
    result.total_trades = total_trades;
    result.profit_factor = calculateProfitFactor(profits, n_profits, losses, n_losses);
    result.calmar_ratio = (result.max_drawdown > 0) ? result.total_return / result.max_drawdown : 0.0;
    result.fitness_score = result.sharpe_ratio * 0.4 + result.total_return * 0.3 + result.win_rate * 0.2 + result.profit_factor * 0.1 - result.max_drawdown * 0.5;
    results[idx] = result;
}

extern "C" void evaluate_population_gpu(
    const CudaStrategyGene* h_genes,
    int population_size,
    const CudaOHLCV* h_data,
    int data_size,
    CudaFitnessResult* h_results
) {
    CudaStrategyGene* d_genes;
    CudaOHLCV* d_data;
    CudaFitnessResult* d_results;
    hipMalloc(&d_genes, population_size * sizeof(CudaStrategyGene));
    hipMalloc(&d_data, data_size * sizeof(CudaOHLCV));
    hipMalloc(&d_results, population_size * sizeof(CudaFitnessResult));
    hipMemcpy(d_genes, h_genes, population_size * sizeof(CudaStrategyGene), hipMemcpyHostToDevice);
    hipMemcpy(d_data, h_data, data_size * sizeof(CudaOHLCV), hipMemcpyHostToDevice);
    int blockSize = 128;
    int numBlocks = (population_size + blockSize - 1) / blockSize;
    evaluate_population_kernel<<<numBlocks, blockSize>>>(d_genes, d_data, population_size, data_size, d_results);
    hipDeviceSynchronize();
    hipMemcpy(h_results, d_results, population_size * sizeof(CudaFitnessResult), hipMemcpyDeviceToHost);
    hipFree(d_genes);
    hipFree(d_data);
    hipFree(d_results);
} 